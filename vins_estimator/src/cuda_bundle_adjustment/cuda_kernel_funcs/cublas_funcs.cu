//
// Created by lmf on 23-10-23.
//

#include "cublas_funcs.cuh"

namespace VINS_FUSION_CUDA_BA {

template<typename T>
std::enable_if_t<(std::is_same<T, double>::value || std::is_same<T, float>::value), hipblasStatus_t>
cublas_gemv_v2(
    const hipblasHandle_t& handle,
    hipblasOperation_t trans,
    int m, int n,
    const T* alpha,
    const T* A, int lda,
    const T* x, int incx,
    const T* beta,
    T* y, int incy
) { hipblasStatus_t cublas_status_local; return cublas_status_local; }
// specialization
template<>
hipblasStatus_t cublas_gemv_v2<double>(
    const hipblasHandle_t& handle,
    hipblasOperation_t trans,
    int m, int n,
    const double* alpha,
    const double* A, int lda,
    const double* x, int incx,
    const double* beta,
    double* y, int incy
) {
    return hipblasDgemv(
        handle,
        trans,
        m,
        n,
        alpha,
        A,
        lda,
        x,
        incx,
        beta,
        y,
        incy
    );
}
template<>
hipblasStatus_t cublas_gemv_v2<float>(
    const hipblasHandle_t& handle,
    hipblasOperation_t trans,
    int m, int n,
    const float* alpha,
    const float* A, int lda,
    const float* x, int incx,
    const float* beta,
    float* y, int incy
) {
    return hipblasSgemv(
        handle,
        trans,
        m,
        n,
        alpha,
        A,
        lda,
        x,
        incx,
        beta,
        y,
        incy
    );
}

// ----------

template<typename T>
std::enable_if_t<(std::is_same<T, double>::value || std::is_same<T, float>::value), hipblasStatus_t>
cublas_gemm_v2(
    const hipblasHandle_t& handle,
    hipblasOperation_t transa, hipblasOperation_t transb,
    int m, int n, int k,
    const T* alpha,
    const T* A, int lda,
    const T* B, int ldb,
    const T* beta,
    T* C, int ldc
) { hipblasStatus_t cublas_status_local; return cublas_status_local; }
// specialization
template<>
hipblasStatus_t cublas_gemm_v2<double>(
    const hipblasHandle_t& handle,
    hipblasOperation_t transa, hipblasOperation_t transb,
    int m, int n, int k,
    const double* alpha,
    const double* A, int lda,
    const double* B, int ldb,
    const double* beta,
    double* C, int ldc
) {
    return hipblasDgemm(
        handle,
        transa, transb,
        m, n, k,
        alpha,
        A, lda,
        B, ldb,
        beta,
        C, ldc
    );
}
template<>
hipblasStatus_t cublas_gemm_v2<float>(
    const hipblasHandle_t& handle,
    hipblasOperation_t transa, hipblasOperation_t transb,
    int m, int n, int k,
    const float* alpha,
    const float* A, int lda,
    const float* B, int ldb,
    const float* beta,
    float* C, int ldc
) {
    return hipblasSgemm(
        handle,
        transa, transb,
        m, n, k,
        alpha,
        A, lda,
        B, ldb,
        beta,
        C, ldc
    );
}

} // namespace VINS_FUSION_CUDA_BA
