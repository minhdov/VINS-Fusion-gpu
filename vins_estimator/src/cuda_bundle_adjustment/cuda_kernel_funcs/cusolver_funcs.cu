//
// Created by lmf on 23-10-23.
//

#include "cusolver_funcs.cuh"

namespace VINS_FUSION_CUDA_BA {

template<typename T>
std::enable_if_t<(std::is_same<T, double>::value || std::is_same<T, float>::value), hipsolverStatus_t>
cusolverDn_potrf_bufferSize(
    const hipsolverHandle_t& handle,
    hipblasFillMode_t uplo,
    int n,
    T* A,
    int lda,
    int* Lwork
) { hipsolverStatus_t cusolver_status_local; return cusolver_status_local; }
// specialization
template<>
hipsolverStatus_t cusolverDn_potrf_bufferSize<double>(
    const hipsolverHandle_t& handle,
    hipblasFillMode_t uplo,
    int n,
    double* A,
    int lda,
    int* Lwork
) {
    return hipsolverDnDpotrf_bufferSize(
        handle,
        uplo,
        n,
        A,
        lda,
        Lwork
    );
}
template<>
hipsolverStatus_t cusolverDn_potrf_bufferSize<float>(
    const hipsolverHandle_t& handle,
    hipblasFillMode_t uplo,
    int n,
    float* A,
    int lda,
    int* Lwork
) {
    return hipsolverDnSpotrf_bufferSize(
        handle,
        uplo,
        n,
        A,
        lda,
        Lwork
    );
}

// ----------

template<typename T>
std::enable_if_t<(std::is_same<T, double>::value || std::is_same<T, float>::value), hipsolverStatus_t>
cusolverDn_potrf(
    const hipsolverHandle_t& handle,
    hipblasFillMode_t uplo,
    int n,
    T* A,
    int lda,
    T* Workspace,
    int Lwork,
    int* devInfo
) { hipsolverStatus_t cusolver_status_local; return cusolver_status_local; }
// specialization
template<>
hipsolverStatus_t cusolverDn_potrf<double>(
    const hipsolverHandle_t& handle,
    hipblasFillMode_t uplo,
    int n,
    double* A,
    int lda,
    double* Workspace,
    int Lwork,
    int* devInfo
) {
    return hipsolverDnDpotrf(
        handle,
        uplo,
        n,
        A,
        lda,
        Workspace,
        Lwork,
        devInfo
    );
}
template<>
hipsolverStatus_t cusolverDn_potrf<float>(
    const hipsolverHandle_t& handle,
    hipblasFillMode_t uplo,
    int n,
    float* A,
    int lda,
    float* Workspace,
    int Lwork,
    int* devInfo
) {
    return hipsolverDnSpotrf(
        handle,
        uplo,
        n,
        A,
        lda,
        Workspace,
        Lwork,
        devInfo
    );
}

// ----------

template<typename T>
std::enable_if_t<(std::is_same<T, double>::value || std::is_same<T, float>::value), hipsolverStatus_t>
cusolverDn_potrs(
    const hipsolverHandle_t& handle,
    hipblasFillMode_t uplo,
    int n,
    int nrhs,
    T* A,
    int lda,
    T* B,
    int ldb,
    int* devInfo
) { hipsolverStatus_t cusolver_status_local; return cusolver_status_local; }
// specialization
template<>
hipsolverStatus_t cusolverDn_potrs<double>(
    const hipsolverHandle_t& handle,
    hipblasFillMode_t uplo,
    int n,
    int nrhs,
    double* A,
    int lda,
    double* B,
    int ldb,
    int* devInfo
) {
    return hipsolverDnDpotrs(
        handle,
        uplo,
        n,
        nrhs,
        A,
        lda,
        B,
        ldb,
        devInfo
    );
}
template<>
hipsolverStatus_t cusolverDn_potrs<float>(
    const hipsolverHandle_t& handle,
    hipblasFillMode_t uplo,
    int n,
    int nrhs,
    float* A,
    int lda,
    float* B,
    int ldb,
    int* devInfo
) {
    return hipsolverDnSpotrs(
        handle,
        uplo,
        n,
        nrhs,
        A,
        lda,
        B,
        ldb,
        devInfo
    );
}

// ------------------------------------------------------------------------------------------------------------------------

template<typename T>
std::enable_if_t<(std::is_same<T, double>::value || std::is_same<T, float>::value), hipsolverStatus_t>
cusolverDn_syevd_bufferSize(
    const hipsolverHandle_t& handle,
    hipsolverEigMode_t jobz,
    hipblasFillMode_t uplo,
    int n,
    T* A,
    int lda,
    T* W,
    int* lwork
) { hipsolverStatus_t cusolver_status_local; return cusolver_status_local; }
// specialization
template<>
hipsolverStatus_t cusolverDn_syevd_bufferSize<double>(
    const hipsolverHandle_t& handle,
    hipsolverEigMode_t jobz,
    hipblasFillMode_t uplo,
    int n,
    double* A,
    int lda,
    double* W,
    int* lwork
) {
    return hipsolverDnDsyevd_bufferSize(
        handle,
        jobz,
        uplo,
        n,
        A,
        lda,
        W,
        lwork
    );
}
template<>
hipsolverStatus_t cusolverDn_syevd_bufferSize<float>(
    const hipsolverHandle_t& handle,
    hipsolverEigMode_t jobz,
    hipblasFillMode_t uplo,
    int n,
    float* A,
    int lda,
    float* W,
    int* lwork
) {
    return hipsolverDnSsyevd_bufferSize(
        handle,
        jobz,
        uplo,
        n,
        A,
        lda,
        W,
        lwork
    );
}

// ----------

template<typename T>
std::enable_if_t<(std::is_same<T, double>::value || std::is_same<T, float>::value), hipsolverStatus_t>
cusolverDn_syevd(
    const hipsolverHandle_t& handle,
    hipsolverEigMode_t jobz,
    hipblasFillMode_t uplo,
    int n,
    T* A,
    int lda,
    T* W,
    T* work,
    int lwork,
    int* devInfo
) { hipsolverStatus_t cusolver_status_local; return cusolver_status_local; }
// specialization
template<>
hipsolverStatus_t cusolverDn_syevd<double>(
    const hipsolverHandle_t& handle,
    hipsolverEigMode_t jobz,
    hipblasFillMode_t uplo,
    int n,
    double* A,
    int lda,
    double* W,
    double* work,
    int lwork,
    int* devInfo
) {
    return hipsolverDnDsyevd(
        handle,
        jobz,
        uplo,
        n,
        A,
        lda,
        W,
        work,
        lwork,
        devInfo
    );
}
template<>
hipsolverStatus_t cusolverDn_syevd<float>(
    const hipsolverHandle_t& handle,
    hipsolverEigMode_t jobz,
    hipblasFillMode_t uplo,
    int n,
    float* A,
    int lda,
    float* W,
    float* work,
    int lwork,
    int* devInfo
) {
    return hipsolverDnSsyevd(
        handle,
        jobz,
        uplo,
        n,
        A,
        lda,
        W,
        work,
        lwork,
        devInfo
    );
}

} // namespace VINS_FUSION_CUDA_BA





