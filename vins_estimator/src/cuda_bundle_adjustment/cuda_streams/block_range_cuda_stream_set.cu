//
// Created by lmf on 23-7-30.
//

#include "block_range_cuda_stream_set.cuh"

namespace VINS_FUSION_CUDA_BA {

BlockRangeStreamSet::BlockRangeStreamSet() {
    hipStreamCreateWithFlags(&cuda_stream_proj_2f1c, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f2c, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_1f2c, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_imu, hipStreamNonBlocking);
}

BlockRangeStreamSet::~BlockRangeStreamSet() {
    SyncAllStream();

    hipStreamDestroy(cuda_stream_proj_2f1c);
    hipStreamDestroy(cuda_stream_proj_2f2c);
    hipStreamDestroy(cuda_stream_proj_1f2c);
    hipStreamDestroy(cuda_stream_imu);
}

void BlockRangeStreamSet::SyncAllStream() const {
    hipStreamSynchronize(cuda_stream_proj_2f1c);
    hipStreamSynchronize(cuda_stream_proj_2f2c);
    hipStreamSynchronize(cuda_stream_proj_1f2c);
    hipStreamSynchronize(cuda_stream_imu);
}

} // namespace VINS_FUSION_CUDA_BA


