//
// Created by lmf on 23-7-26.
//

#include "hessian_rhs_cuda_stream_set.cuh"

namespace VINS_FUSION_CUDA_BA {

HessianRHSStreamSet::HessianRHSStreamSet() {
    hipStreamCreateWithFlags(&cuda_stream_proj_2f1c_hessian_00, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f1c_hessian_01, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f1c_hessian_02, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f1c_hessian_03, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f1c_hessian_04, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f1c_hessian_11, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f1c_hessian_12, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f1c_hessian_13, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f1c_hessian_14, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f1c_hessian_22, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f1c_hessian_23, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f1c_hessian_24, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f1c_hessian_33, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f1c_hessian_34, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f1c_hessian_44, hipStreamNonBlocking);

    hipStreamCreateWithFlags(&cuda_stream_proj_2f1c_rhs_0, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f1c_rhs_1, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f1c_rhs_2, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f1c_rhs_3, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f1c_rhs_4, hipStreamNonBlocking);

    // ----------

    hipStreamCreateWithFlags(&cuda_stream_proj_2f2c_hessian_00, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f2c_hessian_01, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f2c_hessian_02, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f2c_hessian_03, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f2c_hessian_04, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f2c_hessian_05, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f2c_hessian_11, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f2c_hessian_12, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f2c_hessian_13, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f2c_hessian_14, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f2c_hessian_15, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f2c_hessian_22, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f2c_hessian_23, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f2c_hessian_24, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f2c_hessian_25, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f2c_hessian_33, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f2c_hessian_34, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f2c_hessian_35, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f2c_hessian_44, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f2c_hessian_45, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f2c_hessian_55, hipStreamNonBlocking);

    hipStreamCreateWithFlags(&cuda_stream_proj_2f2c_rhs_0, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f2c_rhs_1, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f2c_rhs_2, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f2c_rhs_3, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f2c_rhs_4, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f2c_rhs_5, hipStreamNonBlocking);

    // ----------

    hipStreamCreateWithFlags(&cuda_stream_proj_1f2c_hessian_00, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_1f2c_hessian_01, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_1f2c_hessian_02, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_1f2c_hessian_03, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_1f2c_hessian_11, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_1f2c_hessian_12, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_1f2c_hessian_13, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_1f2c_hessian_22, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_1f2c_hessian_23, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_1f2c_hessian_33, hipStreamNonBlocking);

    hipStreamCreateWithFlags(&cuda_stream_proj_1f2c_rhs_0, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_1f2c_rhs_1, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_1f2c_rhs_2, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_1f2c_rhs_3, hipStreamNonBlocking);

    // ----------

    hipStreamCreateWithFlags(&cuda_stream_imu_hessian_00, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_imu_hessian_01, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_imu_hessian_02, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_imu_hessian_03, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_imu_hessian_11, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_imu_hessian_12, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_imu_hessian_13, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_imu_hessian_22, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_imu_hessian_23, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_imu_hessian_33, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_imu_rhs_0, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_imu_rhs_1, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_imu_rhs_2, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_imu_rhs_3, hipStreamNonBlocking);
}

HessianRHSStreamSet::~HessianRHSStreamSet() {
    SyncAllStream();

    hipStreamDestroy(cuda_stream_proj_2f1c_hessian_00);
    hipStreamDestroy(cuda_stream_proj_2f1c_hessian_01);
    hipStreamDestroy(cuda_stream_proj_2f1c_hessian_02);
    hipStreamDestroy(cuda_stream_proj_2f1c_hessian_03);
    hipStreamDestroy(cuda_stream_proj_2f1c_hessian_04);
    hipStreamDestroy(cuda_stream_proj_2f1c_hessian_11);
    hipStreamDestroy(cuda_stream_proj_2f1c_hessian_12);
    hipStreamDestroy(cuda_stream_proj_2f1c_hessian_13);
    hipStreamDestroy(cuda_stream_proj_2f1c_hessian_14);
    hipStreamDestroy(cuda_stream_proj_2f1c_hessian_22);
    hipStreamDestroy(cuda_stream_proj_2f1c_hessian_23);
    hipStreamDestroy(cuda_stream_proj_2f1c_hessian_24);
    hipStreamDestroy(cuda_stream_proj_2f1c_hessian_33);
    hipStreamDestroy(cuda_stream_proj_2f1c_hessian_34);
    hipStreamDestroy(cuda_stream_proj_2f1c_hessian_44);

    hipStreamDestroy(cuda_stream_proj_2f1c_rhs_0);
    hipStreamDestroy(cuda_stream_proj_2f1c_rhs_1);
    hipStreamDestroy(cuda_stream_proj_2f1c_rhs_2);
    hipStreamDestroy(cuda_stream_proj_2f1c_rhs_3);
    hipStreamDestroy(cuda_stream_proj_2f1c_rhs_4);

    // ----------

    hipStreamDestroy(cuda_stream_proj_2f2c_hessian_00);
    hipStreamDestroy(cuda_stream_proj_2f2c_hessian_01);
    hipStreamDestroy(cuda_stream_proj_2f2c_hessian_02);
    hipStreamDestroy(cuda_stream_proj_2f2c_hessian_03);
    hipStreamDestroy(cuda_stream_proj_2f2c_hessian_04);
    hipStreamDestroy(cuda_stream_proj_2f2c_hessian_05);
    hipStreamDestroy(cuda_stream_proj_2f2c_hessian_11);
    hipStreamDestroy(cuda_stream_proj_2f2c_hessian_12);
    hipStreamDestroy(cuda_stream_proj_2f2c_hessian_13);
    hipStreamDestroy(cuda_stream_proj_2f2c_hessian_14);
    hipStreamDestroy(cuda_stream_proj_2f2c_hessian_15);
    hipStreamDestroy(cuda_stream_proj_2f2c_hessian_22);
    hipStreamDestroy(cuda_stream_proj_2f2c_hessian_23);
    hipStreamDestroy(cuda_stream_proj_2f2c_hessian_24);
    hipStreamDestroy(cuda_stream_proj_2f2c_hessian_25);
    hipStreamDestroy(cuda_stream_proj_2f2c_hessian_33);
    hipStreamDestroy(cuda_stream_proj_2f2c_hessian_34);
    hipStreamDestroy(cuda_stream_proj_2f2c_hessian_35);
    hipStreamDestroy(cuda_stream_proj_2f2c_hessian_44);
    hipStreamDestroy(cuda_stream_proj_2f2c_hessian_45);
    hipStreamDestroy(cuda_stream_proj_2f2c_hessian_55);

    hipStreamDestroy(cuda_stream_proj_2f2c_rhs_0);
    hipStreamDestroy(cuda_stream_proj_2f2c_rhs_1);
    hipStreamDestroy(cuda_stream_proj_2f2c_rhs_2);
    hipStreamDestroy(cuda_stream_proj_2f2c_rhs_3);
    hipStreamDestroy(cuda_stream_proj_2f2c_rhs_4);
    hipStreamDestroy(cuda_stream_proj_2f2c_rhs_5);

    // ----------

    hipStreamDestroy(cuda_stream_proj_1f2c_hessian_00);
    hipStreamDestroy(cuda_stream_proj_1f2c_hessian_01);
    hipStreamDestroy(cuda_stream_proj_1f2c_hessian_02);
    hipStreamDestroy(cuda_stream_proj_1f2c_hessian_03);
    hipStreamDestroy(cuda_stream_proj_1f2c_hessian_11);
    hipStreamDestroy(cuda_stream_proj_1f2c_hessian_12);
    hipStreamDestroy(cuda_stream_proj_1f2c_hessian_13);
    hipStreamDestroy(cuda_stream_proj_1f2c_hessian_22);
    hipStreamDestroy(cuda_stream_proj_1f2c_hessian_23);
    hipStreamDestroy(cuda_stream_proj_1f2c_hessian_33);

    hipStreamDestroy(cuda_stream_proj_1f2c_rhs_0);
    hipStreamDestroy(cuda_stream_proj_1f2c_rhs_1);
    hipStreamDestroy(cuda_stream_proj_1f2c_rhs_2);
    hipStreamDestroy(cuda_stream_proj_1f2c_rhs_3);

    // ----------

    hipStreamDestroy(cuda_stream_imu_hessian_00);
    hipStreamDestroy(cuda_stream_imu_hessian_01);
    hipStreamDestroy(cuda_stream_imu_hessian_02);
    hipStreamDestroy(cuda_stream_imu_hessian_03);
    hipStreamDestroy(cuda_stream_imu_hessian_11);
    hipStreamDestroy(cuda_stream_imu_hessian_12);
    hipStreamDestroy(cuda_stream_imu_hessian_13);
    hipStreamDestroy(cuda_stream_imu_hessian_22);
    hipStreamDestroy(cuda_stream_imu_hessian_23);
    hipStreamDestroy(cuda_stream_imu_hessian_33);
    hipStreamDestroy(cuda_stream_imu_rhs_0);
    hipStreamDestroy(cuda_stream_imu_rhs_1);
    hipStreamDestroy(cuda_stream_imu_rhs_2);
    hipStreamDestroy(cuda_stream_imu_rhs_3);
}

void HessianRHSStreamSet::SyncAllStream() const {
    hipStreamSynchronize(cuda_stream_proj_2f1c_hessian_00);
    hipStreamSynchronize(cuda_stream_proj_2f1c_hessian_01);
    hipStreamSynchronize(cuda_stream_proj_2f1c_hessian_02);
    hipStreamSynchronize(cuda_stream_proj_2f1c_hessian_03);
    hipStreamSynchronize(cuda_stream_proj_2f1c_hessian_04);
    hipStreamSynchronize(cuda_stream_proj_2f1c_hessian_11);
    hipStreamSynchronize(cuda_stream_proj_2f1c_hessian_12);
    hipStreamSynchronize(cuda_stream_proj_2f1c_hessian_13);
    hipStreamSynchronize(cuda_stream_proj_2f1c_hessian_14);
    hipStreamSynchronize(cuda_stream_proj_2f1c_hessian_22);
    hipStreamSynchronize(cuda_stream_proj_2f1c_hessian_23);
    hipStreamSynchronize(cuda_stream_proj_2f1c_hessian_24);
    hipStreamSynchronize(cuda_stream_proj_2f1c_hessian_33);
    hipStreamSynchronize(cuda_stream_proj_2f1c_hessian_34);
    hipStreamSynchronize(cuda_stream_proj_2f1c_hessian_44);

    hipStreamSynchronize(cuda_stream_proj_2f1c_rhs_0);
    hipStreamSynchronize(cuda_stream_proj_2f1c_rhs_1);
    hipStreamSynchronize(cuda_stream_proj_2f1c_rhs_2);
    hipStreamSynchronize(cuda_stream_proj_2f1c_rhs_3);
    hipStreamSynchronize(cuda_stream_proj_2f1c_rhs_4);

    // ----------

    hipStreamSynchronize(cuda_stream_proj_2f2c_hessian_00);
    hipStreamSynchronize(cuda_stream_proj_2f2c_hessian_01);
    hipStreamSynchronize(cuda_stream_proj_2f2c_hessian_02);
    hipStreamSynchronize(cuda_stream_proj_2f2c_hessian_03);
    hipStreamSynchronize(cuda_stream_proj_2f2c_hessian_04);
    hipStreamSynchronize(cuda_stream_proj_2f2c_hessian_05);
    hipStreamSynchronize(cuda_stream_proj_2f2c_hessian_11);
    hipStreamSynchronize(cuda_stream_proj_2f2c_hessian_12);
    hipStreamSynchronize(cuda_stream_proj_2f2c_hessian_13);
    hipStreamSynchronize(cuda_stream_proj_2f2c_hessian_14);
    hipStreamSynchronize(cuda_stream_proj_2f2c_hessian_15);
    hipStreamSynchronize(cuda_stream_proj_2f2c_hessian_22);
    hipStreamSynchronize(cuda_stream_proj_2f2c_hessian_23);
    hipStreamSynchronize(cuda_stream_proj_2f2c_hessian_24);
    hipStreamSynchronize(cuda_stream_proj_2f2c_hessian_25);
    hipStreamSynchronize(cuda_stream_proj_2f2c_hessian_33);
    hipStreamSynchronize(cuda_stream_proj_2f2c_hessian_34);
    hipStreamSynchronize(cuda_stream_proj_2f2c_hessian_35);
    hipStreamSynchronize(cuda_stream_proj_2f2c_hessian_44);
    hipStreamSynchronize(cuda_stream_proj_2f2c_hessian_45);
    hipStreamSynchronize(cuda_stream_proj_2f2c_hessian_55);

    hipStreamSynchronize(cuda_stream_proj_2f2c_rhs_0);
    hipStreamSynchronize(cuda_stream_proj_2f2c_rhs_1);
    hipStreamSynchronize(cuda_stream_proj_2f2c_rhs_2);
    hipStreamSynchronize(cuda_stream_proj_2f2c_rhs_3);
    hipStreamSynchronize(cuda_stream_proj_2f2c_rhs_4);
    hipStreamSynchronize(cuda_stream_proj_2f2c_rhs_5);

    // ----------

    hipStreamSynchronize(cuda_stream_proj_1f2c_hessian_00);
    hipStreamSynchronize(cuda_stream_proj_1f2c_hessian_01);
    hipStreamSynchronize(cuda_stream_proj_1f2c_hessian_02);
    hipStreamSynchronize(cuda_stream_proj_1f2c_hessian_03);
    hipStreamSynchronize(cuda_stream_proj_1f2c_hessian_11);
    hipStreamSynchronize(cuda_stream_proj_1f2c_hessian_12);
    hipStreamSynchronize(cuda_stream_proj_1f2c_hessian_13);
    hipStreamSynchronize(cuda_stream_proj_1f2c_hessian_22);
    hipStreamSynchronize(cuda_stream_proj_1f2c_hessian_23);
    hipStreamSynchronize(cuda_stream_proj_1f2c_hessian_33);

    hipStreamSynchronize(cuda_stream_proj_1f2c_rhs_0);
    hipStreamSynchronize(cuda_stream_proj_1f2c_rhs_1);
    hipStreamSynchronize(cuda_stream_proj_1f2c_rhs_2);
    hipStreamSynchronize(cuda_stream_proj_1f2c_rhs_3);

    // ----------

    hipStreamSynchronize(cuda_stream_imu_hessian_00);
    hipStreamSynchronize(cuda_stream_imu_hessian_01);
    hipStreamSynchronize(cuda_stream_imu_hessian_02);
    hipStreamSynchronize(cuda_stream_imu_hessian_03);
    hipStreamSynchronize(cuda_stream_imu_hessian_11);
    hipStreamSynchronize(cuda_stream_imu_hessian_12);
    hipStreamSynchronize(cuda_stream_imu_hessian_13);
    hipStreamSynchronize(cuda_stream_imu_hessian_22);
    hipStreamSynchronize(cuda_stream_imu_hessian_23);
    hipStreamSynchronize(cuda_stream_imu_hessian_33);
    hipStreamSynchronize(cuda_stream_imu_rhs_0);
    hipStreamSynchronize(cuda_stream_imu_rhs_1);
    hipStreamSynchronize(cuda_stream_imu_rhs_2);
    hipStreamSynchronize(cuda_stream_imu_rhs_3);
}

} // namespace VINS_FUSION_CUDA_BA


