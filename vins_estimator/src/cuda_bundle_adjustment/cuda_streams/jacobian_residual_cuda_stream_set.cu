//
// Created by lmf on 23-7-26.
//

#include "jacobian_residual_cuda_stream_set.cuh"

namespace VINS_FUSION_CUDA_BA {

JacobianResidualStreamSet::JacobianResidualStreamSet() {
    hipStreamCreateWithFlags(&cuda_stream_proj_2f1c_jacobian_0_l, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f1c_jacobian_0_r, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f1c_jacobian_1_l, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f1c_jacobian_1_r, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f1c_jacobian_2_l, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f1c_jacobian_2_r, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f1c_jacobian_3, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f1c_jacobian_4, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f1c_residual, hipStreamNonBlocking);

    hipStreamCreateWithFlags(&cuda_stream_proj_2f2c_jacobian_0_l, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f2c_jacobian_0_r, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f2c_jacobian_1_l, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f2c_jacobian_1_r, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f2c_jacobian_2_l, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f2c_jacobian_2_r, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f2c_jacobian_3_l, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f2c_jacobian_3_r, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f2c_jacobian_4, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f2c_jacobian_5, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f2c_residual, hipStreamNonBlocking);

    hipStreamCreateWithFlags(&cuda_stream_proj_1f2c_jacobian_0_l, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_1f2c_jacobian_0_r, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_1f2c_jacobian_1_l, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_1f2c_jacobian_1_r, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_1f2c_jacobian_2, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_1f2c_jacobian_3, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_1f2c_residual, hipStreamNonBlocking);

    hipStreamCreateWithFlags(&cuda_stream_imu_jacobian_0, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_imu_jacobian_1, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_imu_jacobian_2, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_imu_jacobian_3, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_imu_residual, hipStreamNonBlocking);
}

JacobianResidualStreamSet::~JacobianResidualStreamSet() {
    SyncAllStream();

    hipStreamDestroy(cuda_stream_proj_2f1c_jacobian_0_l);
    hipStreamDestroy(cuda_stream_proj_2f1c_jacobian_0_r);
    hipStreamDestroy(cuda_stream_proj_2f1c_jacobian_1_l);
    hipStreamDestroy(cuda_stream_proj_2f1c_jacobian_1_r);
    hipStreamDestroy(cuda_stream_proj_2f1c_jacobian_2_l);
    hipStreamDestroy(cuda_stream_proj_2f1c_jacobian_2_r);
    hipStreamDestroy(cuda_stream_proj_2f1c_jacobian_3);
    hipStreamDestroy(cuda_stream_proj_2f1c_jacobian_4);
    hipStreamDestroy(cuda_stream_proj_2f1c_residual);

    hipStreamDestroy(cuda_stream_proj_2f2c_jacobian_0_l);
    hipStreamDestroy(cuda_stream_proj_2f2c_jacobian_0_r);
    hipStreamDestroy(cuda_stream_proj_2f2c_jacobian_1_l);
    hipStreamDestroy(cuda_stream_proj_2f2c_jacobian_1_r);
    hipStreamDestroy(cuda_stream_proj_2f2c_jacobian_2_l);
    hipStreamDestroy(cuda_stream_proj_2f2c_jacobian_2_r);
    hipStreamDestroy(cuda_stream_proj_2f2c_jacobian_3_l);
    hipStreamDestroy(cuda_stream_proj_2f2c_jacobian_3_r);
    hipStreamDestroy(cuda_stream_proj_2f2c_jacobian_4);
    hipStreamDestroy(cuda_stream_proj_2f2c_jacobian_5);
    hipStreamDestroy(cuda_stream_proj_2f2c_residual);

    hipStreamDestroy(cuda_stream_proj_1f2c_jacobian_0_l);
    hipStreamDestroy(cuda_stream_proj_1f2c_jacobian_0_r);
    hipStreamDestroy(cuda_stream_proj_1f2c_jacobian_1_l);
    hipStreamDestroy(cuda_stream_proj_1f2c_jacobian_1_r);
    hipStreamDestroy(cuda_stream_proj_1f2c_jacobian_2);
    hipStreamDestroy(cuda_stream_proj_1f2c_jacobian_3);
    hipStreamDestroy(cuda_stream_proj_1f2c_residual);

    hipStreamDestroy(cuda_stream_imu_jacobian_0);
    hipStreamDestroy(cuda_stream_imu_jacobian_1);
    hipStreamDestroy(cuda_stream_imu_jacobian_2);
    hipStreamDestroy(cuda_stream_imu_jacobian_3);
    hipStreamDestroy(cuda_stream_imu_residual);
}

void JacobianResidualStreamSet::SyncAllStream() const {
    hipStreamSynchronize(cuda_stream_proj_2f1c_jacobian_0_l);
    hipStreamSynchronize(cuda_stream_proj_2f1c_jacobian_0_r);
    hipStreamSynchronize(cuda_stream_proj_2f1c_jacobian_1_l);
    hipStreamSynchronize(cuda_stream_proj_2f1c_jacobian_1_r);
    hipStreamSynchronize(cuda_stream_proj_2f1c_jacobian_2_l);
    hipStreamSynchronize(cuda_stream_proj_2f1c_jacobian_2_r);
    hipStreamSynchronize(cuda_stream_proj_2f1c_jacobian_3);
    hipStreamSynchronize(cuda_stream_proj_2f1c_jacobian_4);
    hipStreamSynchronize(cuda_stream_proj_2f1c_residual);

    hipStreamSynchronize(cuda_stream_proj_2f2c_jacobian_0_l);
    hipStreamSynchronize(cuda_stream_proj_2f2c_jacobian_0_r);
    hipStreamSynchronize(cuda_stream_proj_2f2c_jacobian_1_l);
    hipStreamSynchronize(cuda_stream_proj_2f2c_jacobian_1_r);
    hipStreamSynchronize(cuda_stream_proj_2f2c_jacobian_2_l);
    hipStreamSynchronize(cuda_stream_proj_2f2c_jacobian_2_r);
    hipStreamSynchronize(cuda_stream_proj_2f2c_jacobian_3_l);
    hipStreamSynchronize(cuda_stream_proj_2f2c_jacobian_3_r);
    hipStreamSynchronize(cuda_stream_proj_2f2c_jacobian_4);
    hipStreamSynchronize(cuda_stream_proj_2f2c_jacobian_5);
    hipStreamSynchronize(cuda_stream_proj_2f2c_residual);

    hipStreamSynchronize(cuda_stream_proj_1f2c_jacobian_0_l);
    hipStreamSynchronize(cuda_stream_proj_1f2c_jacobian_0_r);
    hipStreamSynchronize(cuda_stream_proj_1f2c_jacobian_1_l);
    hipStreamSynchronize(cuda_stream_proj_1f2c_jacobian_1_r);
    hipStreamSynchronize(cuda_stream_proj_1f2c_jacobian_2);
    hipStreamSynchronize(cuda_stream_proj_1f2c_jacobian_3);
    hipStreamSynchronize(cuda_stream_proj_1f2c_residual);

    hipStreamSynchronize(cuda_stream_imu_jacobian_0);
    hipStreamSynchronize(cuda_stream_imu_jacobian_1);
    hipStreamSynchronize(cuda_stream_imu_jacobian_2);
    hipStreamSynchronize(cuda_stream_imu_jacobian_3);
    hipStreamSynchronize(cuda_stream_imu_residual);
}

} // namespace VINS_FUSION_CUDA_BA

