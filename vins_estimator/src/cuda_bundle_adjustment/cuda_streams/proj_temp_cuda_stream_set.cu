//
// Created by lmf on 23-7-31.
//

#include "proj_temp_cuda_stream_set.cuh"

namespace VINS_FUSION_CUDA_BA {

ProjTempStreamSet::ProjTempStreamSet() {
    hipStreamCreateWithFlags(&proj_2f1c_temp, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&proj_2f2c_temp, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&proj_1f2c_temp, hipStreamNonBlocking);
}

ProjTempStreamSet::~ProjTempStreamSet() {
    SyncAllStream();

    hipStreamDestroy(proj_2f1c_temp);
    hipStreamDestroy(proj_2f2c_temp);
    hipStreamDestroy(proj_1f2c_temp);
}

void ProjTempStreamSet::SyncAllStream() const {
    hipStreamSynchronize(proj_2f1c_temp);
    hipStreamSynchronize(proj_2f2c_temp);
    hipStreamSynchronize(proj_1f2c_temp);
}

} // namespace VINS_FUSION_CUDA_BA

