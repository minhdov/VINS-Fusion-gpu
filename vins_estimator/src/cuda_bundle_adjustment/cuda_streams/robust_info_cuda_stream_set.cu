//
// Created by lmf on 23-7-26.
//

#include "robust_info_cuda_stream_set.cuh"

namespace VINS_FUSION_CUDA_BA {

RobustInfoStreamSet::RobustInfoStreamSet() {
    hipStreamCreateWithFlags(&cuda_stream_proj_2f1c_robust_info, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f2c_robust_info, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_1f2c_robust_info, hipStreamNonBlocking);

    hipStreamCreateWithFlags(&cuda_stream_imu_robust_info, hipStreamNonBlocking);
}

RobustInfoStreamSet::~RobustInfoStreamSet() {
    SyncAllStream();

    hipStreamDestroy(cuda_stream_proj_2f1c_robust_info);
    hipStreamDestroy(cuda_stream_proj_2f2c_robust_info);
    hipStreamDestroy(cuda_stream_proj_1f2c_robust_info);

    hipStreamDestroy(cuda_stream_imu_robust_info);
}

void RobustInfoStreamSet::SyncAllStream() const {
    hipStreamSynchronize(cuda_stream_proj_2f1c_robust_info);
    hipStreamSynchronize(cuda_stream_proj_2f2c_robust_info);
    hipStreamSynchronize(cuda_stream_proj_1f2c_robust_info);

    hipStreamSynchronize(cuda_stream_imu_robust_info);
}

} // namespace VINS_FUSION_CUDA_BA