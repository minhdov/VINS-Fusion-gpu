//
// Created by lmf on 23-10-30.
//

#include "update_cuda_stream_set.cuh"

namespace VINS_FUSION_CUDA_BA {

UpdateStreamSet::UpdateStreamSet() {
    hipStreamCreateWithFlags(&cuda_stream_proj_2f1c_update, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_2f2c_update, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&cuda_stream_proj_1f2c_update, hipStreamNonBlocking);

    hipStreamCreateWithFlags(&cuda_stream_imu_update, hipStreamNonBlocking);
}

UpdateStreamSet::~UpdateStreamSet() {
    SyncAllStream();

    hipStreamDestroy(cuda_stream_proj_2f1c_update);
    hipStreamDestroy(cuda_stream_proj_2f2c_update);
    hipStreamDestroy(cuda_stream_proj_1f2c_update);

    hipStreamDestroy(cuda_stream_imu_update);
}

void UpdateStreamSet::SyncAllStream() const {
    hipStreamSynchronize(cuda_stream_proj_2f1c_update);
    hipStreamSynchronize(cuda_stream_proj_2f2c_update);
    hipStreamSynchronize(cuda_stream_proj_1f2c_update);

    hipStreamSynchronize(cuda_stream_imu_update);
}

} // namespace VINS_FUSION_CUDA_BA

